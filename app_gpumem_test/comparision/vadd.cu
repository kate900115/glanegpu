
#include <hip/hip_runtime.h>
#include <stdio.h>
// for time measurement
#include <chrono>
#include <ctime>
#include <iostream>

__global__ void vecAdd(int m, int n, float* A,  float* C ){
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
//	printf("A[%d][%d]\n",i,j);
	if ((i<m)&&(j<n)) {
		C[i*n+j] = A[i*n+j]/7;
	//	printf("A[%d][%d]=%f\n",i,j,A[i*n+j]);
	}
}

int main(){
	int m = 64;
	int n = 64;
	float* h_a = NULL;
	float* h_b = NULL;
	float* h_c = NULL;
	float* d_a = NULL;
	float* d_b = NULL;
	float* d_c = NULL;
	h_a = (float*)malloc(m*n*sizeof(float));
	h_b = (float*)malloc(m*n*sizeof(float));
	h_c = (float*)malloc(m*n*sizeof(float));

	hipMalloc((void**)&d_a, m*n*sizeof(float));
	hipMalloc((void**)&d_b, m*n*sizeof(float));
	hipMalloc((void**)&d_c, m*n*sizeof(float));

	if ((h_a==NULL)||(h_b==NULL)||(h_c==NULL)||(d_a==NULL)||(d_b==NULL)||(d_c==NULL)){
		printf("cannot allocate memory.\n");
	}
	
	//memset(h_c,0,m*n*sizeof(float));
	for (int i=0; i<m; i++){
		for (int j=0; j<n; j++){
			h_a[i*n+j]=i+j;
			h_b[i*n+j]=i+j;
			h_c[i*n+j]=0;
		//	printf("%f,%f,%f\n",h_a[i*n+j],h_b[i*n+j],h_c[i*n+j]);
		}
	}

	int count = 0;
	dim3 grid((n+15)/16, (m+15)/16,1);
	dim3 block(16, 16,1);
	
	auto start = std::chrono::high_resolution_clock::now();
	while (count<100){	
		hipMemcpy(d_a, h_a, m*n*sizeof(float), hipMemcpyHostToDevice);
		//cudaMemcpy(d_b, h_b, m*n*sizeof(float), cudaMemcpyHostToDevice);
		hipMemcpy(d_c, h_c, m*n*sizeof(float), hipMemcpyHostToDevice);

		vecAdd<<<grid, block>>>	(m, n, d_a, d_c);
		hipMemcpy(h_c, d_c, m*n*sizeof(float), hipMemcpyDeviceToHost);
		count++;
	}
	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> diff = end - start;
	std::cout<<"it took me "<<diff.count()<<" seconds."<<std::endl;
	
	return 0;		
}
