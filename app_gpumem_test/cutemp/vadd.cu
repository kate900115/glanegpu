#include "hip/hip_runtime.h"
#include "../GlaneGPUstack.h"

// virtual address on GPU
__device__ void* inBuf;
__device__ void* outBuf;
__device__ void* AQueue;
__device__ void* requestBuf;

// physical address on GPU
__device__ unsigned long p_inBuf;
__device__ unsigned long p_outBuf;
__device__ unsigned long p_AQueue;
__device__ unsigned long p_reqBuf;
__device__ int kernelID;

// cursor of AQueue
__device__ int cursor;


__device__ void CUDAkernelInitialization(void* dptr){

	// initialize AQ and cursor
	struct AQentry* AQ = (struct AQentry*) dptr;
	AQueue = (void*) AQ;
	for (int i=0; i<16; i++){
		AQ[i].isInUse = 0;
		AQ[i].MemFreelistIdx = i;
	}
	cursor = 0;
	printf("initialization finished!\n");

	// initialize request buffer
	requestBuf = dptr + AQsize * sizeof (struct AQentry);
	struct reqBuf* requestBuffer = (struct reqBuf*) requestBuf;
	requestBuffer->isInUse = false;		

	// initialize inBuf & outBuf
	inBuf = requestBuf + sizeof (struct reqBuf);
	outBuf = inBuf + 2 * MemBufferSize * m * n * sizeof (float);	
}

__device__ void AQmoveCursor(){
	if (cursor !=15){
		cursor++;
	}
	else{
		cursor = 0;
	}
	printf("cursor = %d\n", cursor);	
	struct AQentry* AQ = (struct AQentry*) AQueue;

	// to check wait until the next AQ entry is available
	while (AQ[cursor].isInUse);
}

__device__ void pushRequest(){
	struct reqBuf* requestBuffer = (struct reqBuf*) requestBuf;
	while (requestBuffer->isInUse);
}

	
extern "C" __global__ void vadd(int* virtualAddr, int* FPGAreqBuf, struct physAddr* addrPacket){
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int count = 0;
	
	if ((i==0)&&(j==0)){
		CUDAkernelInitialization((void*)virtualAddr+100*sizeof(int));
		printf("GPU side address = %p\n",addrPacket->dptrPhyAddrOnGPU);
		printf("kernel ID = %d\n", addrPacket->kernelID);
	}
	__syncthreads();
	float* c = (float*)outBuf;
	float* a = (float*)inBuf;	
	//__syncthreads();

	while(count<100){
		count++;
		if ((i==0)&&(j==0)){
			while (*virtualAddr!=0){
				atomicCAS(virtualAddr, 0,0);
			}
		}

		// CUDA kernel execution
		if ((i<m)&&(j<n)) {
			c[i*n+j] = a[i*n+j] + i + j;
		}

		__syncthreads();

		if ((i==0)&&(j==0)){
			atomicCAS(virtualAddr, 0, 1);
			//printf("GPU: lock is set to be 1\n");
		}

		__syncthreads();

		if ((i==0)&&(j==0)){
			*FPGAreqBuf = 0;
			//printf("GPU: flag is set to be 0\n");
			AQmoveCursor();
		}
	}
}


