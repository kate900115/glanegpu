#include "hip/hip_runtime.h"
#include "../GlaneGPUstack.h"

__device__ void* inBuf;
__device__ void* outBuf;
__device__ void* AQueue;
__device__ void* requestBuf;

__device__ int cursor;

__device__ void CUDAkernelInitialization(void* dptr){

	// initialize AQ and cursor
	struct AQentry* AQ = (struct AQentry*) dptr;
	AQueue = (void*) AQ;
	for (int i=0; i<16; i++){
		AQ[i].isInUse = 0;
		AQ[i].MemFreelistIdx = i;
	}
	cursor = 0;
	printf("initialization finished!\n");

	// initialize request buffer
	requestBuf = dptr + AQsize * sizeof (struct AQentry);
	struct reqBuf* requestBuffer = (struct reqBuf*) requestBuf;
	requestBuffer->isInUse = false;		

	// initialize inBuf & outBuf
	inBuf = requestBuf + sizeof (struct reqBuf);
	outBuf = inBuf + 2 * MemBufferSize * m * n * sizeof (float);	
}

__device__ void AQmoveCursor(){
	if (cursor !=15){
		cursor++;
	}
	else{
		cursor = 0;
	}
	printf("cursor = %d\n", cursor);	
	struct AQentry* AQ = (struct AQentry*) AQueue;

	// to check wait until the next AQ entry is available
	while (AQ[cursor].isInUse);
}

__device__ void pushRequest(){
	struct reqBuf* requestBuffer = (struct reqBuf*) requestBuf;
	while (requestBuffer->isInUse);
}

	
extern "C" __global__ void vadd(int* d_lock, int* flag, struct physAddr* addrPacket){
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int count = 0;
	
	if ((i==0)&&(j==0)){
		CUDAkernelInitialization((void*)d_lock+100*sizeof(int));
	}
	__syncthreads();
	float* c = (float*)outBuf;
	float* a = (float*)inBuf;	
	//__syncthreads();

	while(count<100){
		count++;
		if ((i==0)&&(j==0)){
			while (*d_lock!=0){
				atomicCAS(d_lock, 0,0);
			}
		}

		// CUDA kernel execution
		if ((i<m)&&(j<n)) {
			c[i*n+j] = a[i*n+j] + i + j;
		}

		__syncthreads();

		if ((i==0)&&(j==0)){
			atomicCAS(d_lock, 0, 1);
			//printf("GPU: lock is set to be 1\n");
		}

		__syncthreads();

		if ((i==0)&&(j==0)){
			*flag = 0;
			//printf("GPU: flag is set to be 0\n");
			AQmoveCursor();
		}
	}
}

